#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hiprand.h>
#include <assert.h>
#include <unistd.h>
#include <hipblas.h>
#include <iostream>
#include <complex.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cublas_beamformer.h"

using namespace std;

// CUDA-specific function prototypes
void print_matrix(const hipComplex *A, int nr_rows_A, int nr_cols_A, int nr_sheets_A);

void print_matrix2(const float *A, int nr_rows_A, int nr_cols_A);

void GPU_fill(hipComplex *A, int nr_rows_A, int nr_cols_A);

void GPU_fill2(hipComplex *A, int nr_rows_A, int nr_cols_A);

__global__
void data_restructure(signed char * data, hipComplex * data_restruc);

void beamform();

__global__
void sti_reduction(hipComplex * data_in, float * data_out);


// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill(hipComplex *A, int nr_rows_A, int nr_cols_A) {
	hipComplex *G;
	G = new hipComplex[nr_rows_A*nr_cols_A];
	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
		G[i].x = (i + 1)%(nr_rows_A*nr_cols_A/(BN_BIN));
		G[i].y = (i + 1)%(nr_rows_A*nr_cols_A/(BN_BIN));

	}

	hipMemcpy(A,G,nr_rows_A * nr_cols_A * sizeof(hipComplex),hipMemcpyHostToDevice);
	delete[] G;
}

void GPU_fill2(hipComplex *A, int nr_rows_A, int nr_cols_A) {
	hipComplex *G;
	G = new hipComplex[nr_rows_A*nr_cols_A];
	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
		G[i].x = i%(nr_rows_A*nr_cols_A/(BN_BIN));
		G[i].y = i%(nr_rows_A*nr_cols_A/(BN_BIN));
	}

	hipMemcpy(A,G,nr_rows_A * nr_cols_A * sizeof(hipComplex),hipMemcpyHostToDevice);
	delete[] G;
}

void print_matrix(const hipComplex *A, int nr_rows_A, int nr_cols_A, int nr_sheets_A) {
	for(int i = 0; i < nr_rows_A; ++i){
		for(int j = 0; j < nr_cols_A; ++j){
			for(int k = 0; k < nr_sheets_A; ++k){
				//				cout << A[j * nr_rows_A + i].x << "+" << A[j * nr_rows_A + i].y << "i" <<" ";
				printf("%i,%i,%i: %e + %e i\n",i,j,k,A[k*nr_rows_A*nr_cols_A + j * nr_rows_A + i].x, A[k*nr_rows_A*nr_cols_A + j * nr_rows_A + i].y);
			}
		}
		//			cout << endl;
	}
	//		cout << endl;
	//	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
	//		printf("%i,: %e + %e i\n",i,A[i].x, A[i].y);
	//	}
}


void print_matrix2(const float *A, int nr_rows_A, int nr_cols_A) {
	//	for(int j = 0; j < nr_cols_A; ++j){
	//		for(int i = 0; i < nr_rows_A; ++i){
	//			//cout << A[j * nr_rows_A + i].x << "+" << A[j * nr_rows_A + i].y << "i" <<" ";
	//			printf("%i,%i: %e\n",i,j,A[j * nr_rows_A + i]);
	//		}
	//		cout << endl;
	//	}
	//	cout << endl;

	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
		printf("%i,: %e\n",i,A[i]);
	}
}

// Struct defintion for beamformer metadata
typedef struct bf_metadata_struct {
	float offsets[14];
	char cal_filename[65];
	char algorithm[65];
	char weight_filename[65];
	long long unsigned int xid;
} bf_metadata;
static bf_metadata my_metadata;

static hipComplex * d_weights = NULL;
void update_weights(char * filename){
	printf("In update_weights()...\n");
	char weight_filename[128];
	strcpy(weight_filename, filename);
	FILE * weights;
	float * bf_weights;
	float complex * weights_dc;
	float complex * weights_dc_n;

	// Allocate heap memory for file data
	bf_weights = (float *)malloc(2*BN_WEIGHTS*sizeof(float));
	weights_dc = (float complex *)malloc(BN_WEIGHTS*sizeof(float complex *));
	weights_dc_n = (float complex *)malloc(BN_WEIGHTS*sizeof(float complex *));
	weights = fopen(weight_filename, "r");

	int j;
	if (weights != NULL) {
		fread(bf_weights, sizeof(float), 2*BN_WEIGHTS, weights);

		fread(my_metadata.offsets, sizeof(float), 14, weights);
		fread(my_metadata.cal_filename, sizeof(char), 64, weights);
		my_metadata.cal_filename[64] = '\0';
		fread(my_metadata.algorithm, sizeof(char), 64, weights);
		my_metadata.algorithm[64] = '\0';
		fread(&(my_metadata.xid), sizeof(long long unsigned int), 1, weights);

		// Extract all path information from weight_filename for metadata
		char * short_filename = strrchr(weight_filename, '/');
		if (short_filename != NULL) {
			strcpy(my_metadata.weight_filename, short_filename+1);
		}
		else {
			strcpy(my_metadata.weight_filename, weight_filename);
		}



		// Convert to complex numbers (do a conjugate at the same time)
		for(j = 0; j < BN_WEIGHTS; j++){
			weights_dc_n[j] = bf_weights[2*j] - bf_weights[(2*j)+1]*I;
		}

		// Transpose the weights
		int m,n;
		float complex transpose[BN_BEAM][BN_ELE_BLOC*BN_BIN];
		for(m=0;m<BN_BEAM;m++){
			for(n=0;n<BN_ELE_BLOC*BN_BIN;n++){
				transpose[m][n] = weights_dc_n[m*BN_ELE_BLOC*BN_BIN + n];
			}
		}
		for(n=0;n<BN_ELE_BLOC*BN_BIN;n++){
			for(m=0;m<BN_BEAM;m++){
				weights_dc[n*BN_BEAM+ m] = transpose[m][n];
			}
		}
		fclose(weights);
	}
	free(bf_weights);



	// Copy weights to device
	hipMemcpy(d_weights, weights_dc, BN_WEIGHTS*sizeof(hipComplex), hipMemcpyHostToDevice); //r_weights instead of weights_dc //*BN_TIME

	free(weights_dc);
}

void bf_get_offsets(float * offsets){
	for(int i = 0; i<BN_BEAM; i++){
		offsets[i] = my_metadata.offsets[i];
	}
}

void bf_get_cal_filename(char * cal_filename){
	for(int i = 0; i< 65; i++){
		cal_filename[i] = my_metadata.cal_filename[i];
	}
}

void bf_get_algorithm(char * algorithm){
	for(int i = 0; i< 65; i++){
		algorithm[i] = my_metadata.algorithm[i];
	}
}

void bf_get_weight_filename(char * weight_filename){
	int num_chars = strlen(my_metadata.weight_filename);
	for (int i = 0; i < num_chars; i++) {
		weight_filename[i] = my_metadata.weight_filename[i];
	}
	for (int i = num_chars; i < 64; i++) {
		weight_filename[i] = ' ';
	}
	weight_filename[64] = '\0';
}

long long unsigned int bf_get_xid(){
	return my_metadata.xid;
}

static hipComplex **d_arr_A = NULL; static hipComplex **d_arr_B = NULL; static hipComplex **d_arr_C = NULL;
static hipComplex * d_beamformed = NULL;
static hipComplex * d_data = NULL;
static signed char * d_data1 = NULL; // Device memory for input data
static float * d_outputs;

static hipblasHandle_t handle;
void init_beamformer(){
	// Allocate memory for the weights, data, beamformer output, and sti output.

	hipMalloc((void **)&d_weights, BN_WEIGHTS*sizeof(hipComplex)); //*BN_TIME

	hipMalloc((void **)&d_data1, 2*BN_SAMP*sizeof(signed char));

	hipMalloc((void **)&d_data, BN_SAMP*sizeof(hipComplex));

	hipError_t err_malloc = hipMalloc((void **)&d_beamformed, BN_TBF*sizeof(hipComplex));
	if (err_malloc != hipSuccess) {
		printf("CUDA Error (cudaMalloc2): %s\n", hipGetErrorString(err_malloc));
	}

	hipMalloc((void **)&d_outputs, BN_POL*(BN_OUTPUTS*sizeof(float)/2));

        /**********************************************************
         * Create a handle for CUBLAS
         **********************************************************/
        hipblasCreate(&handle);

	// This is all memory allocated to arrays that are used by gemmBatched.
	// Allocate 3 arrays on CPU
	hipError_t cudaStat;

	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	nr_rows_A = BN_BEAM;
	nr_cols_A = BN_ELE_BLOC;
	nr_rows_B = BN_ELE_BLOC;
	nr_cols_B = BN_TIME;
	nr_rows_C = BN_BEAM;
	nr_cols_C = BN_TIME;

	// Allocate memory to host arrays.
	const hipComplex **h_arr_A = 0; const hipComplex **h_arr_B = 0; hipComplex **h_arr_C = 0;
	h_arr_A = (const hipComplex **)malloc(nr_rows_A * nr_cols_A *BN_BIN*sizeof(const hipComplex*));
	h_arr_B = (const hipComplex **)malloc(nr_rows_B * nr_cols_B *BN_BIN*sizeof(const hipComplex*));
	h_arr_C = (hipComplex **)malloc(nr_rows_C * nr_cols_C *BN_BIN*sizeof(hipComplex*));

	// Allocate memory for each batch in an array.
	for(int i = 0; i < BN_BIN; i++){
		h_arr_A[i] = d_weights + i*nr_rows_A*nr_cols_A;
		h_arr_B[i] = d_data + i*nr_rows_B*nr_cols_B;
		h_arr_C[i] = d_beamformed + i*nr_rows_C*nr_cols_C;
	}

	//	delete[] d_A;
	//	delete[] d_B;

	// Allocate memory to arrays on device.
	cudaStat = hipMalloc((void **)&d_arr_A,nr_rows_A * nr_cols_A * BN_BIN * sizeof(hipComplex*));
	assert(!cudaStat);
	cudaStat = hipMalloc((void **)&d_arr_B,nr_rows_B * nr_cols_B * BN_BIN * sizeof(hipComplex*));
	assert(!cudaStat);
	cudaStat = hipMalloc((void **)&d_arr_C,nr_rows_C * nr_cols_C * BN_BIN * sizeof(hipComplex*));
	assert(!cudaStat);

	// Copy memory from host to device.
	cudaStat = hipMemcpy(d_arr_A,h_arr_A,nr_rows_A * nr_cols_A * BN_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice);
	assert(!cudaStat);
	cudaStat = hipMemcpy(d_arr_B,h_arr_B,nr_rows_B * nr_cols_B * BN_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice);
	assert(!cudaStat);
	cudaStat = hipMemcpy(d_arr_C,h_arr_C,nr_rows_C * nr_cols_C * BN_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice);
	assert(!cudaStat);

        
}

__global__
void data_restructure(signed char * data, hipComplex * data_restruc){

	int e = threadIdx.x;
	int t = blockIdx.x;
	int f = blockIdx.y;

	//Restructure data so that the frequency bin is the slowest moving index
	data_restruc[f*BN_TIME*BN_ELE_BLOC + t*BN_ELE_BLOC + e].x = data[2*(t*BN_BIN*BN_ELE_BLOC + f*BN_ELE_BLOC + e)]*1.0f;
	data_restruc[f*BN_TIME*BN_ELE_BLOC + t*BN_ELE_BLOC + e].y = data[2*(t*BN_BIN*BN_ELE_BLOC + f*BN_ELE_BLOC + e) + 1]*1.0f;
}

signed char * data_in(char * input_filename){
	FILE * data;

	// File data pointers
	signed char * bf_data;

	// Complex data pointers
	// float complex * data_dc;

	// Allocate heap memory for file data
	bf_data = (signed char *)malloc(2*BN_SAMP*sizeof(signed char));
	//data_dc = (float complex *)malloc(BN_SAMP*sizeof(float complex *));

	// Open files
	data = fopen(input_filename, "r");

	/*********************************************************
	 * Read in Data
	 *********************************************************/
	if (data != NULL) {
		fread(bf_data, sizeof(signed char), 2*BN_SAMP, data);
		/*
		int j;
		// Make 'em complex!
		for (j = 0; j < BN_SAMP; j++) {
			data_dc[j] = bf_data[2*j] + bf_data[(2*j)+1]*I;
		}
		*/

		// Specify grid and block dimensions
		// dim3 dimBlock_d(BN_ELE, 1, 1);
		// dim3 dimGrid_d(BN_TIME, BN_BIN, 1);

		//hipComplex * d_data_in = d_data1;
		//hipComplex * d_data_out = d_data;

		//hipMemcpy(d_data_in,    data_dc,   BN_SAMP*sizeof(hipComplex), hipMemcpyHostToDevice);

		// Restructure data for hipblasCgemmBatched function.
		//data_restructure<<<dimGrid_d, dimBlock_d>>>(d_data_in, d_data_out);

		fclose(data);
	}
	//free(bf_data);
	//free(data_dc);
	return bf_data;
}

void beamform() {
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C;
	nr_rows_A = BN_BEAM;
	nr_cols_A = BN_ELE_BLOC;
	nr_rows_B = BN_ELE_BLOC;
	nr_cols_B = BN_TIME;
	nr_rows_C = BN_BEAM;

	// Leading dimensions are always the rows of each matrix since the data is stored in a column-wise order.
	int lda=nr_rows_A,ldb=nr_rows_B,ldc=nr_rows_C;
	hipComplex alf;
	hipComplex bet;

	alf.x = 1;
	alf.y = 0;
	bet.x = 0;
	bet.y = 0;

	int batchCount = BN_BIN; 				// There must be the same number of batches in each array.

	hipblasStatus_t stat;
	/*
		This function performs a matrix multiplication of the data and the weights.
		Weights - d_arr_A, Data - d_arr_B, and the output - d_arr_C.
	*/
	stat = hipblasCgemmBatched(
			handle,							// handle to the cuBLAS library context.
			HIPBLAS_OP_N,					// Operation on matrices within array A.
			HIPBLAS_OP_N,					// Operation on matrices within array B.
			nr_rows_A,						// Number of rows in matrix A and C.
			nr_cols_B,						// Number of columns in matrix B and C.
			nr_cols_A,						// Number of columns and rows in matrix A and B respectively.
			&alf,							// Scalar used for multiplication.
			(const hipComplex **)d_arr_A,	// Weight array of pointers.
			lda,							// Leading dimension of each batch or matrix in array A.
			(const hipComplex **)d_arr_B,	// Data array of pointers.
			ldb,							// Leading dimension of each batch or matrix in array B.
			&bet,							// Scalar used for multiplication.
			(hipComplex **)d_arr_C,			// Output array of pointers.
			ldc,							// Leading dimension of each batch or matrix in array C.
			batchCount);					// Number of batches in each array.


	if(stat != HIPBLAS_STATUS_SUCCESS){
		cerr << "hipblasCgemmBatched failed" << endl;
		exit(1);
	}
	assert(!hipGetLastError());

	//Free GPU memory
	//	hipFree(d_A);
	//	hipFree(d_B);
	//	hipFree(d_C);

	// Destroy the handle
	//hipblasDestroy(handle);

}

__global__
void sti_reduction(hipComplex * data_in, float * data_out) {

	int f = blockIdx.x;
	int b = blockIdx.y;
	int t = threadIdx.x;
	int s = blockIdx.z;

	int h = sample_idx(s*BN_TIME_STI + t,b,f);						// Preprocessor macro used for the output of the beamformer. More detail can be seen in the header file. (First set of beams)
	int h1 = sample_idx(s*BN_TIME_STI + t,b+BN_BEAM1,f);				// Preprocessor macro used for the output of the beamformer. More detail can be seen in the header file. (Last set of beams)

	// Temporary variables used for updating.
	float beam_power1;
	float beam_power2;
	float cross_power1;
	float cross_power2;

	hipFloatComplex samp1;
	hipFloatComplex samp2;
	float scale = 1.0/BN_TIME_STI; 									// Scale power by number of samples per STI window.

	__shared__ hipFloatComplex reduced_array1[BN_STI_BLOC];
	__shared__ hipFloatComplex reduced_array[BN_STI_BLOC];

	if (t < BN_TIME_STI) {
		// X polarization (XX*).
		samp1.x = data_in[h].x;
		samp1.y = data_in[h].y;
		beam_power1 = (samp1.x * samp1.x) + (samp1.y * samp1.y);	// Beamformer output multiplied by its conjugate (absolute value squared).
		reduced_array[t].x = beam_power1;

		// Y polarization (YY*).
		samp2.x = data_in[h1].x;
		samp2.y = data_in[h1].y;
		beam_power2 = (samp2.x * samp2.x) + (samp2.y * samp2.y);	// Beamformer output multiplied by its conjugate (absolute value squared).
		reduced_array[t].y = beam_power2;

		// Cross polarization (XY*).
		cross_power1 = (samp1.x * samp2.x) + (samp1.y * samp2.y);	// Real part of cross polarization.
		cross_power2 = (samp1.y * samp2.x) - (samp1.x * samp2.y);	// Imaginary part of cross polarization.
		reduced_array1[t].x = cross_power1;
		reduced_array1[t].y = cross_power2;
	}
	else{
		reduced_array[t].x = 0.0;
		reduced_array[t].y = 0.0;
		reduced_array1[t].x = 0.0;
		reduced_array1[t].y = 0.0;
	}
	__syncthreads();

	// Reduction is performed by splitting up the threads in each block and summing them all up.
	// The number of threads in each block needs to be a power of two in order for the reduction to work. (No left over threads).
	for(int k = blockDim.x/2; k>0; k>>=1){
		if(t<k){
			reduced_array[t].x += reduced_array[t+k].x;
			reduced_array[t].y += reduced_array[t+k].y;
			reduced_array1[t].x += reduced_array1[t+k].x;
			reduced_array1[t].y += reduced_array1[t+k].y;
		}
		__syncthreads();
	}

	// After reduction is complete, assign each reduced to value to appropriate position in output array.
	if(t == 0){
		data_out[output_idx(0,b,s,f)] = reduced_array[0].x*scale; 	// XX*.
		data_out[output_idx(1,b,s,f)] = reduced_array[0].y*scale; 	// YY*.
		data_out[output_idx(2,b,s,f)] = reduced_array1[0].x*scale; 	// XY* real.
		data_out[output_idx(3,b,s,f)] = reduced_array1[0].y*scale;	// XY* imaginary.
	}
}

void run_beamformer(signed char * data_in, float * data_out) {
	// Specify grid and block dimensions
	dim3 dimBlock(BN_STI_BLOC, 1, 1);
	dim3 dimGrid(BN_BIN, BN_BEAM1, BN_STI);

	// Specify grid and block dimensions
	dim3 dimBlock_d(BN_ELE_BLOC, 1, 1);
	dim3 dimGrid_d(BN_TIME, BN_BIN, 1);

	signed char * d_restruct_in = d_data1;
	hipComplex * d_restruct_out = d_data;

	hipMemcpy(d_restruct_in, data_in, 2*BN_SAMP*sizeof(signed char), hipMemcpyHostToDevice);

	// Restructure data for hipblasCgemmBatched function.
	data_restructure<<<dimGrid_d, dimBlock_d>>>(d_restruct_in, d_restruct_out);

//	printf("Starting beamformer\n");

	// Call beamformer function containing hipblasCgemmBatched()
	beamform();
	hipError_t err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("CUDA Error (beamform): %s\n", hipGetErrorString(err_code));
	}

	hipComplex * d_sti_in = d_beamformed;
	float * d_sti_out = d_outputs;

//	printf("Starting sti_reduction\n");

	// Call STI reduction kernel.
	sti_reduction<<<dimGrid, dimBlock>>>(d_sti_in, d_sti_out);

//	printf("Finishing sti_reduction\n");

	err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("CUDA Error (sti_reduction): %s\n", hipGetErrorString(err_code));
	}

	// Copy output data from device to host.
	hipMemcpy(data_out, d_sti_out, BN_POL*(BN_OUTPUTS*sizeof(float)/2),hipMemcpyDeviceToHost);

	// hipFree(d_data);
	// hipFree(d_outputs);
}


void rtbfCleanup() {
	// Free up GPU memory at the end of a program
	if (d_beamformed != NULL) {
		hipFree(d_beamformed);
	}

	if (d_data != NULL) {
		hipFree(d_data);
	}

	if (d_data1 != NULL) {
		hipFree(d_data1);
	}

	if (d_outputs != NULL) {
		hipFree(d_outputs);
	}

	if (d_weights != NULL) {
		hipFree(d_weights);
	}

	if (d_arr_A != NULL) {
		hipFree(d_arr_A);
	}

	if (d_arr_B != NULL) {
		hipFree(d_arr_B);
	}

	if (d_arr_C != NULL) {
		hipFree(d_arr_C);
	}
	// Free up and release cublas handle
	hipblasDestroy(handle);
}
